#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>

#include "utils.hxx"

#define NUM_THREADS 256

#define CUDA_assert(expr) do {                                                    \
	hipError_t code = expr;                                                      \
	if (code != hipSuccess) {                                                    \
		std::cerr << "Line: " << __LINE__ << " | " << #expr << std::endl;         \
		std::cerr << "    CUDA error: " << hipGetErrorString(code) << std::endl; \
		return false;                                                             \
	}                                                                             \
} while (0)

#define MAX_THREADS 512
#define MAX_BLOCKS 32768

static __global__ void general_swap(int* data, int block_size, int pass, int div) {
	bitonic_swap(data, blockIdx.y * MAX_THREADS + threadIdx.x, blockIdx.x * div + threadIdx.y, block_size, pass);
}

static __global__ void remain_swap(int* data, int block_size, int pass, int number_of_blocks) {
	bitonic_swap(data, blockIdx.x * MAX_THREADS + threadIdx.x, number_of_blocks, block_size, pass);
}

static __global__ void remain_of_remain_swap(int* data, int block_size, int pass, int number_of_blocks, int offset) {
	bitonic_swap(data, offset + threadIdx.x, number_of_blocks, block_size, pass);
}

bool gpu_bitonic_sort(std::vector<int>& data) {
	int* device_data;

	CUDA_assert(hipMalloc(&device_data, data.size() * sizeof(int)));
	CUDA_assert(hipMemcpy(device_data, data.data(), data.size() * sizeof(int), hipMemcpyHostToDevice));

	int number_of_stages = get_number_of_stages(data);

	for (int stage = 1; stage <= number_of_stages; stage++) {
		int block_size = 1 << stage;
		int number_of_blocks = data.size() >> stage;
		int remain_size = data.size() & (block_size - 1);

		for (int pass = 0; pass < stage; pass++) {
			int step = block_size >> 1;

			dim3 number_of_cuda_blocks(number_of_blocks, max(1, step / MAX_THREADS));
			dim3 number_of_cuda_threads(min(MAX_THREADS, step), 1);
			int div = 1;

			if (number_of_cuda_blocks.x * number_of_cuda_blocks.y > MAX_BLOCKS) {
				div = MAX_THREADS / number_of_cuda_threads.x;
				number_of_cuda_blocks.x /= div;
				number_of_cuda_threads.y = div;
			}

			general_swap<<<number_of_cuda_blocks, number_of_cuda_threads>>>(device_data, block_size, pass, div);
			std::cout << number_of_cuda_blocks.x << ' ' << number_of_cuda_blocks.y << ' ' << number_of_cuda_threads.x << ' ' << number_of_cuda_threads.y << std::endl;
			CUDA_assert(hipGetLastError());

			if (remain_size != 0) {
				int threads = remain_size - step;
				if (threads > 0) {
					int count = threads / MAX_THREADS;
					remain_swap<<<count, MAX_THREADS>>>(device_data, block_size, pass, number_of_blocks);
					CUDA_assert(hipGetLastError());
					remain_of_remain_swap<<<1, threads % MAX_THREADS>>>(device_data, block_size, pass, number_of_blocks, count * MAX_THREADS);
					CUDA_assert(hipGetLastError());
				}
			}

			CUDA_assert(hipDeviceSynchronize());

			block_size >>= 1;
			number_of_blocks <<= 1;
		}
	}

	CUDA_assert(hipMemcpy(data.data(), device_data, data.size() * sizeof(int), hipMemcpyDeviceToHost));
	CUDA_assert(hipFree(device_data));

	return true;
}
