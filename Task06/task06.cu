#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>
#include <iostream>

const int BLOCK_NUM = 1024;

#pragma region CPU
void bitonic_merge(int *a, int low, int cnt, int dir)
{
	if (cnt > 1)
	{
		int k = cnt / 2;
		for (int i = low; i < low + k; i++)
		{
			if (dir == (a[i] > a[i + k]))
			{
				std::swap(a[i], a[i + k]);
			}
		}
		bitonic_merge(a, low, k, dir);
		bitonic_merge(a, low + k, k, dir);
	}
}

void bitonic_sort_cpu(int *a, int low, int cnt, int dir)
{
	if (cnt > 1)
	{
		int k = cnt / 2;
		bitonic_sort_cpu(a, low, k, 1);
		bitonic_sort_cpu(a, low + k, k, 0);

		bitonic_merge(a, low, cnt, dir);
	}
}
#pragma endregion CPU

#pragma region GPU
#define GPUERRCHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void bitonic_sort_step(int *a, int j, int k) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int ixj = tid ^ j;

	if (ixj > tid)
	{
		if ((tid & k) == 0)
		{
			if (a[tid] > a[ixj])
			{
				int temp;
				temp = a[tid];
				a[tid] = a[ixj];
				a[ixj] = temp;
			}
		}
		else
			if (a[tid] < a[ixj])
			{
				int temp;
				temp = a[tid];
				a[tid] = a[ixj];
				a[ixj] = temp;
			}
		}
}

void bitonic_sort_gpu(int *arr, int cnt) {
	int *a;
	GPUERRCHK(hipMalloc((void**)&a, cnt * sizeof(int)));
	GPUERRCHK(hipMemcpy(a, arr, cnt * sizeof(int), hipMemcpyHostToDevice));

	dim3 blocks = (cnt < BLOCK_NUM) ? 1 : BLOCK_NUM;;
	dim3 threads = (cnt < BLOCK_NUM) ? cnt : cnt / BLOCK_NUM;;

	for (unsigned int k = 2; k <= cnt; k *= 2)
	{
		for (unsigned int j = k / 2; j > 0; j /= 2)
		{
			bitonic_sort_step <<<blocks, threads >> > (a, j, k);
		}
	}
	GPUERRCHK(hipMemcpy(arr, a, cnt * sizeof(int), hipMemcpyDeviceToHost));
	hipFree(a);

}
#pragma endregion GPU

bool test(int *arr, int cnt)
{
	bool is_sorted = true;
	for (int i = 0; i < cnt; i++)
	{
		if (arr[i - 1] > arr[i])
		{
			is_sorted = false;
		}
	}

	if (is_sorted) {
		std::cout << "Array is sorted" << std::endl;
		return 0;
	}
	else
	{
		std::cout << "Array is not sorted" << std::endl;
		return 1;
	}
}

int main()
{
	int cnt = 0;
	char mode;
	const int n = pow(2, 18);
	int *arr = (int*)malloc(1e9 * sizeof(int));
	for (cnt = 0; cnt < n; cnt++)
	{
		arr[cnt] = rand() % 10000;
	}

	std::cin >> mode;
	if (mode == 'c')
	{
		auto begin_cpu = std::chrono::steady_clock::now();
		bitonic_sort_cpu(arr, 0, cnt, 1);
		auto end_cpu = std::chrono::steady_clock::now();

		auto elapsed_ms_cpu = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - begin_cpu);
		std::cout << "The cpu_time: " << elapsed_ms_cpu.count() << " ms\n";

		test(arr, cnt);
	}

	if (mode == 'g')
	{
		auto begin_gpu = std::chrono::steady_clock::now();
		bitonic_sort_gpu(arr, cnt);
		auto end_gpu = std::chrono::steady_clock::now();

		auto elapsed_ms_gpu = std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - begin_gpu);
		std::cout << "The gpu_time: " << elapsed_ms_gpu.count() << " ms\n";

		test(arr, cnt);
	}
    return 0;
}
