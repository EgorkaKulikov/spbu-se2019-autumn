#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>


#include <cstdio>
#include <random>
#include <time.h>
#include <iostream>

using namespace std;

const int THREADS_NUM = 1024;
const int EXPRMNT_NUM = 10;

void bitonic_sort(int* arr, int size) 
{
  int num_stages = 0;
  //log2 size floored
  for (int i = size; i > 0; i >>= 1, num_stages++);
  num_stages--;

  for (int stage = 1; stage <= num_stages; stage++) 
  {
    int num_passes = stage;
    int block_size = 1 << stage;
    int num_blocks = size >> stage;
    for (int pass = 0; pass < num_passes; pass++) 
    {
      int step = block_size >> 1;
      for (int block = 0; block < num_blocks; block++) 
      {
        bool ascending = ((block >> pass) & 1) == 0;
        for (int i = 0; i < step; i++) 
        {
          int index = block * block_size + i;
          if ((ascending && (arr[index] > arr[index + step])) 
          || (!ascending && (arr[index] < arr[index + step]))) 
          {
            int tmp = arr[index];
            arr[index] = arr[index + step];
            arr[index + step] = tmp;
          }
        }
      }
      block_size >>= 1;
      num_blocks <<= 1;
    }
  }
  cout << endl;
}

__device__
void swap_gpu(int* array, int first, int second)
{   
    int tmp = array[first];
    array[first] = array[second];
    array[second] = tmp;
}

__global__
void bitonic_exchange_gpu(int* dev_values, int block_size, unsigned long stage)
{
    unsigned int i, match; 

    i = threadIdx.x + blockDim.x * blockIdx.x;

    match = i + (block_size >> 1);

    bool ascending_i = (i & (1 << stage)) == 0;
    bool ascending_m = (match & (1 << stage)) == 0;

    if (ascending_i ^ ascending_m == 1) return;
    
    if ((dev_values[i] > dev_values[match]) == ascending_i)
    {
      swap_gpu(dev_values, i, match);
    }
}

void bitonic_sort_CUDA(int* arr, int size) 
{
  int* cudarr;
  if (hipMalloc(&cudarr, size * sizeof(int)) != hipSuccess) 
  {
    cerr << "Error when allocating device memory" << endl;
    exit(7);
  }

  if (hipMemcpy(cudarr, arr, size * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) 
  {
    cerr << "Error when copying memory" << endl;
    exit(8);
  }

  dim3 threadsPerBlock = (size < THREADS_NUM) ? 1 : THREADS_NUM;
  dim3 numCUDABlocks = (size < THREADS_NUM) ? size : size / THREADS_NUM;

  int num_stages = -1;
  for (int i = size; i > 0; i >>= 1, num_stages++);

  for (int stage = 1; stage <= num_stages; stage++) 
  {
    int num_passes = stage;
    int block_size = 1 << stage;
    for (int pass = 1; pass <= num_passes; pass++) 
    {
      bitonic_exchange_gpu <<< numCUDABlocks, threadsPerBlock >>> (cudarr, block_size, stage);
      hipError_t errSync = hipGetLastError();
      hipError_t errAsync = hipDeviceSynchronize();
      if (errSync != hipSuccess && errAsync != hipSuccess) 
      {
        cerr << "CUDA execution error" << endl;
        exit(9);
      }
      block_size >>= 1;
    }
  }

  if (hipMemcpy(arr, cudarr, size * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) 
  {
    cerr << "Error when copying memory" << endl;
    exit(8);
  }

  hipFree(cudarr);
}

int main(int argc, char **argv) 
{
  if (argc != 3) 
  {
    cerr << "Wrong number of arguments" << endl;
    exit(1);
  }

  FILE* file = fopen(argv[1], "r");
  if (file == nullptr) 
  {
    cerr << "No such file of directory" << endl;
    exit(2);
  }

  int size;
  if (fscanf(file, "%d", &size) != 1) 
  {
    cerr << "Wrong file format" << endl;
    exit(3);
  }

  if (size == 0) 
  {
    return 0;
  }

  int power_of_2_size = 1;
  while (power_of_2_size < size) 
  {
    power_of_2_size <<= 1;
  }

  int* data = new int[power_of_2_size];
  if (data == nullptr) 
  {
    cerr << "Memory allocation error" << endl;
    exit(4);
  }

  int max_value = (1 << 30);
  for (int i = 0; i < size; i++) 
  {
    if (fscanf(file, "%d", &(data[i])) != 1) 
    {
      cerr << "Error while reading file!" << endl;
      return 3;
    }
    max_value = max(max_value, data[i]);
  }

  fclose(file);

  for (int i = size; i < power_of_2_size; i++) 
  {
    data[i] = max_value;
  }

  double time_avg = 0.0;
  for (int counter = 0; counter < EXPRMNT_NUM; counter++)
  {
    int* data_copy = (int*) malloc(power_of_2_size * sizeof(int));
    memcpy(data_copy, data, power_of_2_size * sizeof(int));
    double time_count = -1.0;
    if (argv[2][0] == 's') 
    {
      double timestamp = clock();
      bitonic_sort(data_copy, power_of_2_size);
      time_count = (double) (clock() - timestamp);
    }
    else if (argv[2][0] == 'p') 
    {
      double timestamp = clock();
      bitonic_sort_CUDA(data_copy, power_of_2_size);
      time_count = (double) (clock() - timestamp);
    }
    else 
    {
      cerr << "Wrong launch parameter" << endl;
      exit(5);
    }

    int prev = data_copy[0];
    //check if sorted
    for (int i = 1; i < power_of_2_size; i++) 
    {
      if (data_copy[i] < prev) 
      {
        cerr << "error at element: " << i << "data size: " << size << "changed size: " << power_of_2_size << "prev value: " << prev << "next value: " << data_copy[i] << endl;
      }
      prev = data_copy[i];
    }
    time_avg += time_count;
    free(data_copy);
  }
  cout << "Time: " << (time_avg / EXPRMNT_NUM) / CLOCKS_PER_SEC << "s, " << size << " elements" << endl;
  delete[] data;
  return 0;
}