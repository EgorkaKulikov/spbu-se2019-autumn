
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <iomanip>
#include <cstring>

using namespace std;

const int THREADS_PER_BLOCK = 1024;

class hidden_markov_model {

    double **transitions;
    double **emissions;
    double *initial_distribution;
    int *observations;
    double **alpha;
    double **beta;

public:
    int n_states;
    int n_observations;
    int observations_dict_size;

    hidden_markov_model(int n_states, int n_observations, int observations_dict_size);
    void generate_random_parameters();
    void generate_random_observations(int *dict);
    void print();

    double run_forward_algo();
    void run_viterbi_algo(int *hidden_states_idxs);
    void run_baum_welch_algo(int epochs);

    double run_GPU_forward_algo();
    void run_GPU_baum_welch_algo(int epochs);
};


hidden_markov_model::hidden_markov_model(int _n_states, int _n_observations, int _observations_dict_size) {
    n_states = _n_states;
    n_observations = _n_observations;
    observations_dict_size = _observations_dict_size;

    transitions = new double *[n_states];
    for (int i = 0; i < n_states; i++) {
        transitions[i] = new double [n_states];
    }
    emissions = new double *[n_states];
    for (int i = 0; i < n_states; i++) {
        emissions[i] = new double [observations_dict_size];
    }
    initial_distribution = new double [n_states];
    observations = new int [n_observations];

    alpha = new double *[n_observations];
    for (int i = 0; i < n_observations; i++) {
        alpha[i] = new double [n_states];
    }
    beta = new double *[n_observations];
    for (int i = 0; i < n_observations; i++) {
        beta[i] = new double [n_states];
    }
}


void hidden_markov_model::generate_random_parameters() {
    random_device device;
    mt19937 gen(device());
    uniform_int_distribution<> distribution(0, 1000);
    // Generate stochastic matrix A
    for (int i = 0; i < n_states; ++i) {
        double sum = 0;
        for (int j = 0; j < n_states; ++j) {
            transitions[i][j] = distribution(gen);
            sum += transitions[i][j];
        }
        for (int j = 0; j < n_states; ++j) {
            transitions[i][j] /= sum;
        }
    }
    // Generate stochastic matrix B
    for (int i = 0; i < n_states; ++i) {
        double sum = 0;
        for (int j = 0; j < observations_dict_size; ++j) {
            emissions[i][j] = distribution(gen);
            sum += emissions[i][j];
        }
        for (int j = 0; j < observations_dict_size; ++j) {
            emissions[i][j] /= sum;
        }
    }
    // Generate initial probabilities distribution (pi)
    double sum = 0;
    for (int i = 0; i < n_states; ++i) {
        initial_distribution[i] = distribution(gen);
        sum += initial_distribution[i];
    }
    for (int i = 0; i < n_states; ++i) {
        initial_distribution[i] /= sum;
    }
}


void hidden_markov_model::generate_random_observations(int *dict) {
    random_device device;
    mt19937 gen(device());
    uniform_int_distribution<> distribution(0, observations_dict_size - 1);
    for (int i = 0; i < n_observations; ++i) {
        observations[i] = dict[distribution(gen)];
    }
}


void hidden_markov_model::print() {
    cout << "Transitions matrix: \n";
    for (int i = 0; i < n_states; ++i) {
        for (int j = 0; j < n_states; ++j) {
            cout << transitions[i][j] << " ";
        }
        cout << endl;
    }
    cout << "Emissions matrix: \n";
    for (int i = 0; i < n_states; ++i) {
        for (int j = 0; j < observations_dict_size; ++j) {
            cout << emissions[i][j] << " ";
        }
        cout << endl;
    }
    cout << "Initial distribution: \n";
    for (int i = 0; i < n_states; ++i) {
        cout << initial_distribution[i] << " ";
    }
    cout << endl;
    cout << "Observations: \n";
    for (int i = 0; i < n_observations; ++i) {
        cout << observations[i] << " ";
    }
    cout << endl;
}


double hidden_markov_model::run_forward_algo() {
    // alpha[t][i] is the probability of all observations up to time t in the hidden state i
    for (int i = 0; i < n_states; ++i) {
        alpha[0][i] = initial_distribution[i] * emissions[i][observations[0]];
    }
    for (int t = 1; t < n_observations; ++t) {
        for (int i = 0; i < n_states; ++i) {
            alpha[t][i] = 0.0;
            for (int j = 0; j < n_states; ++j) {
                alpha[t][i] += alpha[t - 1][j] * transitions[j][i] * emissions[i][observations[t]];
            }
        }
    }
    double likelihood = 0.0;
    for (int i = 0; i < n_states; ++i) {
        likelihood += alpha[n_observations - 1][i];
    }
    return likelihood;
}


void hidden_markov_model::run_viterbi_algo(int *hidden_states_idxs) {
    // alpha[t][i] is the most likely probability of all observations up to time [t] in the hidden state [i]
    for (int i = 0; i < n_states; ++i) {
        alpha[0][i] = initial_distribution[i] * emissions[i][observations[0]];
    }
    int backtrack[n_observations][n_states];
    for (int t = 1; t <= n_observations; ++t) {
        for (int i = 0; i < n_states; ++i) {
            if (t != n_observations) {
                alpha[t][i] = -1;
                for (int j = 0; j < n_states; ++j) {
                    double prob = transitions[j][i] * emissions[i][observations[t]];
                    if (alpha[t][i] < alpha[t - 1][j] * prob) {
                        alpha[t][i] = alpha[t - 1][j] * prob;
                    }
                }
            }
            double temp = -1;
            for (int j = 0; j < n_states; ++j) {
                if (temp < alpha[t - 1][j] * transitions[j][i]) {
                    temp = alpha[t - 1][j] * transitions[j][i];
                    backtrack[t - 1][i] = j;
                }
            }
        }
    }
    // Calculate the most likely last hidden state
    double max_likelihood = -1;
    for (int i = 0; i < n_states; ++i) {
        if (alpha[n_observations - 1][i] > max_likelihood) {
            max_likelihood = alpha[n_observations - 1][i];
            hidden_states_idxs[n_observations - 1] = i;
        }
    }
    // Go backtracking
    for (int i = n_observations - 2; i >= 0; --i) {
        int next_state = hidden_states_idxs[i + 1];
        hidden_states_idxs[i] = backtrack[i + 1][next_state];
    }
}


void hidden_markov_model::run_baum_welch_algo(int epochs) {
    generate_random_parameters();
    // Allocate memory
    auto **gamma = new double *[n_observations];
    auto **xi = new double *[n_states];
    for (int i = 0; i < n_observations; ++i) {
        gamma[i] = new double[n_states];
    }
    for (int i = 0; i < n_states; ++i) {
        xi[i] = new double[n_states];
    }
    // Fit the model
    for (int epoch = 0; epoch < epochs; ++epoch) {
        double likelihood = run_forward_algo();
        /*if (epoch % 5 == 0) {
            cout << "Epoch: " << epoch << " | Likelihood: " << likelihood << endl;
        }*/
        // Calculate probabilities: beta, gamma and xi
        for (int i = 0; i < n_states; ++i) {
            beta[n_observations - 1][i] = 1;
            gamma[n_observations - 1][i] = (alpha[n_observations - 1][i] * beta[n_observations - 1][i]) / likelihood;
            memset(xi[i], 0.0, n_states * sizeof(double));
        }
        for (int t = n_observations - 2; t >= 0; --t) {
            for (int i = 0; i < n_states; ++i) {
                beta[t][i] = 0.0;
                for (int j = 0; j < n_states; ++j) {
                    double prob = transitions[i][j] * emissions[j][observations[t + 1]];
                    beta[t][i] += beta[t + 1][j] * prob;
                    xi[i][j] += (alpha[t][i] * prob * beta[t + 1][j]) / likelihood;
                }
                gamma[t][i] = (alpha[t][i] * beta[t][i]) / likelihood;
            }
        }
        // Optimize parameters
        for (int i = 0; i < n_states; ++i) {
            double occupation_prob = 0.0;
            for (int t = 0; t < n_observations - 1; ++t) {
                occupation_prob += gamma[t][i];
            }
            for (int j = 0; j < n_states; ++j) {
                transitions[i][j] = xi[i][j] / occupation_prob;
            }
            double total_occupation_prob = occupation_prob + gamma[n_observations - 1][i];
            for (int j = 0; j < n_observations; ++j) {
                double sum = 0.0;
                for (int t = 0; t < n_observations; ++t) {
                    sum += (observations[t] == observations[j]) ? gamma[t][i] : 0;
                }
                emissions[i][observations[j]] = sum / total_occupation_prob;
            }
            initial_distribution[i] = gamma[0][i];
        }
    }
    // Free memory
    for (int i = 0; i < n_observations; ++i) {
        delete gamma[i];
    }
    delete[] gamma;
    for (int i = 0; i < n_states; ++i) {
        delete xi[i];
    }
    delete[] xi;
}


__global__
void make_forward_step(double *alpha, double *transitions, double *emissions, int *observations,
                       int t, int n_states, int observations_dict_size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_states) {
        alpha[t * n_states + i] = 0.0;
        for (int j = 0; j < n_states; ++j) {
            alpha[t * n_states + i] += alpha[(t - 1) * n_states + j] * transitions[j * n_states + i] * emissions[i * observations_dict_size + observations[t]];
        }
    }
}


template <typename T>
T *allocate_matrix_on_device(int rows, int cols){
    T *matrix;
    hipMalloc(&matrix, sizeof(T) * rows * cols);
    return matrix;
}


template <typename T>
void copy_matrix_to_device(T *d_matrix, T **matrix, int rows, int cols) {
    T **temp_matrix = new T *[rows * cols];
    for (int i = 0; i < rows; ++i)
        memcpy(temp_matrix + i * cols, matrix[i], cols * sizeof(T));
    hipMemcpy(d_matrix, temp_matrix, rows * cols * sizeof(T), hipMemcpyHostToDevice);
    delete[] temp_matrix;
}


template <typename T>
void copy_matrix_from_device(T **matrix, T *d_matrix, int rows, int cols) {
    T **temp_matrix = new T *[rows * cols];
    hipMemcpy(temp_matrix, d_matrix, rows * cols * sizeof(T), hipMemcpyDeviceToHost);
    for (int i = 0; i < rows; ++i) {
        memcpy(matrix[i], temp_matrix + i * cols, cols * sizeof(T));
    }
    delete[] temp_matrix;
}


double hidden_markov_model::run_GPU_forward_algo() {
    for (int i = 0; i < n_states; ++i) {
        alpha[0][i] = initial_distribution[i] * emissions[i][observations[0]];
    }

    double *d_alpha = allocate_matrix_on_device<double>(n_observations, n_states);
    double *d_transitions = allocate_matrix_on_device<double>(n_states, n_states);
    double *d_emissions = allocate_matrix_on_device<double>(n_states, observations_dict_size);
    int *d_observations;
    hipMalloc(&d_observations, n_observations * sizeof(int));

    copy_matrix_to_device<double>(d_alpha, alpha, n_observations, n_states);
    copy_matrix_to_device<double>(d_transitions, transitions, n_states, n_states);
    copy_matrix_to_device<double>(d_emissions, emissions, n_states, observations_dict_size);
    hipMemcpy(d_observations, observations, n_observations * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    for (int t = 1; t < n_observations; ++t) {
        make_forward_step<<<n_states/THREADS_PER_BLOCK+1, THREADS_PER_BLOCK>>>(
                d_alpha, d_transitions, d_emissions, d_observations,
                t, n_states, observations_dict_size);
        hipDeviceSynchronize();
    }

    copy_matrix_from_device<double>(alpha, d_alpha, n_observations, n_states);

    double likelihood = 0.0;
    for (int i = 0; i < n_states; ++i) {
        likelihood += alpha[n_observations - 1][i];
    }

    hipFree(d_alpha);
    hipFree(d_transitions);
    hipFree(d_emissions);
    hipFree(d_observations);

    return likelihood;
}


void hidden_markov_model::run_GPU_baum_welch_algo(int epochs) {
    generate_random_parameters();
    // Allocate memory
    auto **gamma = new double *[n_observations];
    auto **xi = new double *[n_states];
    for (int i = 0; i < n_observations; ++i) {
        gamma[i] = new double[n_states];
    }
    for (int i = 0; i < n_states; ++i) {
        xi[i] = new double[n_states];
    }
    // Fit the model
    for (int epoch = 0; epoch < epochs; ++epoch) {
        double likelihood = run_GPU_forward_algo();
        /*if (epoch % 5 == 0) {
            cout << "Epoch: " << epoch << " | Likelihood: " << likelihood << endl;
        }*/
        // Calculate probabilities: beta, gamma and xi
        for (int i = 0; i < n_states; ++i) {
            beta[n_observations - 1][i] = 1;
            gamma[n_observations - 1][i] = (alpha[n_observations - 1][i] * beta[n_observations - 1][i]) / likelihood;
            memset(xi[i], 0.0, n_states * sizeof(double));
        }
        for (int t = n_observations - 2; t >= 0; --t) {
            for (int i = 0; i < n_states; ++i) {
                beta[t][i] = 0.0;
                for (int j = 0; j < n_states; ++j) {
                    double prob = transitions[i][j] * emissions[j][observations[t + 1]];
                    beta[t][i] += beta[t + 1][j] * prob;
                    xi[i][j] += (alpha[t][i] * prob * beta[t + 1][j]) / likelihood;
                }
                gamma[t][i] = (alpha[t][i] * beta[t][i]) / likelihood;
            }
        }
        // Optimize parameters
        for (int i = 0; i < n_states; ++i) {
            double occupation_prob = 0.0;
            for (int t = 0; t < n_observations - 1; ++t) {
                occupation_prob += gamma[t][i];
            }
            for (int j = 0; j < n_states; ++j) {
                transitions[i][j] = xi[i][j] / occupation_prob;
            }
            double total_occupation_prob = occupation_prob + gamma[n_observations - 1][i];
            for (int j = 0; j < n_observations; ++j) {
                double sum = 0.0;
                for (int t = 0; t < n_observations; ++t) {
                    sum += (observations[t] == observations[j]) ? gamma[t][i] : 0;
                }
                emissions[i][observations[j]] = sum / total_occupation_prob;
            }
            initial_distribution[i] = gamma[0][i];
        }
    }
    // Free memory
    for (int i = 0; i < n_observations; ++i) {
        delete gamma[i];
    }
    delete[] gamma;
    for (int i = 0; i < n_states; ++i) {
        delete xi[i];
    }
    delete[] xi;
}


int main() {
    //cout << fixed << setprecision(7);
    for (int n_states = 10; n_states < 10000; n_states *= 2) {
        const int n_observations = 300;
        const int observations_dict_size = 2;
        const int epochs = 100;
        clock_t start_t, end_t;

        // Create observations dictionary
        int dict[observations_dict_size];
        for (int i = 0; i < observations_dict_size; ++i) {
            dict[i] = i;
        }
        // Create HMM
        hidden_markov_model hmm(n_states, n_observations, observations_dict_size);
        hmm.generate_random_parameters();
        hmm.generate_random_observations(dict);
        //hmm.print();
        //cout << "States: " << n_states << " | Observations: " << n_observations << endl << endl;
        cout << "[" << n_states << ", ";

        start_t = clock();
        hmm.run_GPU_baum_welch_algo(epochs);
        end_t = clock();
        cout << double(end_t - start_t) / CLOCKS_PER_SEC << ", ";
        //cout << "Likelihood: " << likelihood << " | ";
        //cout << "Time elapsed on GPU: " << double(end_t - start_t) / CLOCKS_PER_SEC << endl << endl;

        start_t = clock();
        hmm.run_baum_welch_algo(epochs);
        end_t = clock();
        cout << double(end_t - start_t) / CLOCKS_PER_SEC << "], " << endl;
        //cout << "Likelihood: " << likelihood << " | ";
        //cout << "Time elapsed on CPU: " << double(end_t - start_t) / CLOCKS_PER_SEC << endl << endl;
    }

/*
    int hidden[n_observations];
    start_t = clock();
    hmm.run_viterbi_algo(hidden);
    end_t = clock();
    cout << "Hidden states (Viterbi algorithm) calculated." << endl;
    cout << "Time elapsed: " << double(end_t - start_t) / CLOCKS_PER_SEC << endl << endl;

    cout << "Fitting HMM (Baum-Welch algorithm):" << endl;
    start_t = clock();
    hmm.run_baum_welch_algo(epochs);
    end_t = clock();
    cout << "Time elapsed: " << double(end_t - start_t) / CLOCKS_PER_SEC << endl << endl;
*/
    return 0;
}
