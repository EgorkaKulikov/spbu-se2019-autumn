#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#include "string.h"
#include "bitonic_sorts.cuh"
#include "utils.cuh"

#define AMOUNT_MEASUREMENTS = 20

int main(char* args)
{
    clock_t start;
    unsigned long upperBound = 1024 << 13;
    
    for (unsigned size = 1024; size <= upperBound; size <<= 1)
    {        
        srand(time(NULL));
        
        size_t size_mem_array = size*sizeof(int);

        int* array = (int*) malloc(size_mem_array);
        int* temp_array = (int*) malloc(size_mem_array);
        
        double timeGPU = 0;
        double timeCPU = 0;

        generate_random_array(array, size);
        
        for (int i = 0; i < 20; i++)
        {
            memcpy(temp_array, array, size_mem_array);
            
            start = clock();
            bitonic_sort(temp_array, size);
            timeCPU += ((double) (clock() - start)) / CLOCKS_PER_SEC;

            memcpy(temp_array, array, size_mem_array);

            start = clock();
            bitonic_sort_gpu(temp_array, size);
            timeGPU += ((double) (clock() - start)) / CLOCKS_PER_SEC;
        }

        printf("%ld %f %f\n", size, timeGPU / AMOUNT_MEASUREMENTS, timeCPU / AMOUNT_MEASUREMENTS);
        
        free(array);
        free(temp_array);
    }

    return 0;
}