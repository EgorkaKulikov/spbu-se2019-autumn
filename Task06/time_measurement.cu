#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#include "string.h"
#include "bitonic_sorts.cuh"
#include "utils.cuh"

int main(char* args)
{
    clock_t start;
    unsigned long upperBound = 1024 << 13;
    
    for (unsigned size = 1024; size <= upperBound; size <<= 1)
    {        
        srand(time(NULL));
        
        size_t size_mem_array = size*sizeof(int);

        int* array = (int*) malloc(size_mem_array);
        int* temp_array = (int*) malloc(size_mem_array);
        
        double timeGPU = 0;
        double timeCPU = 0;

        generate_random_array(array, size);
        
        for (int i = 0; i < 20; i++)
        {
            
            memcpy(temp_array, array, size_mem_array);
            
            start = clock();
            bitonic_sort(temp_array, size);
            timeCPU += (((double) (clock() - start)) / CLOCKS_PER_SEC) / 20;

            memcpy(temp_array, array, size_mem_array);

            start = clock();
            bitonic_sort_gpu(temp_array, size);
            timeGPU += (((double) (clock() - start)) / CLOCKS_PER_SEC) / 20;
        }

        printf("%ld %f %f\n", size, timeGPU, timeCPU);
        
        free(array);
        free(temp_array);
    }

    return 0;
}